
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define BLOCK_SIZE 16

__global__ void gpu_matrix_mult(int *a, int *b, int *c, int m, int n, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if (col < k && row < m) {
        for (int i = 0; i < n; i++) {
            sum += a[row * n + i] * b[i * k + col];
        }
        c[row * k + col] = sum;
    }
}

__global__ void gpu_square_matrix_mult(int *d_a, int *d_b, int *d_result, int n) {
    __shared__ int tile_a[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ int tile_b[BLOCK_SIZE][BLOCK_SIZE];

    int row = blockIdx.y * BLOCK_SIZE + threadIdx.y;
    int col = blockIdx.x * BLOCK_SIZE + threadIdx.x;
    int tmp = 0;
    int idx;

    for (int sub = 0; sub < gridDim.x; ++sub) {
        idx = row * n + sub * BLOCK_SIZE + threadIdx.x;
        tile_a[threadIdx.y][threadIdx.x] = (idx >= n * n) ? 0 : d_a[idx];

        idx = (sub * BLOCK_SIZE + threadIdx.y) * n + col;
        tile_b[threadIdx.y][threadIdx.x] = (idx >= n * n) ? 0 : d_b[idx];
        __syncthreads();

        for (int k = 0; k < BLOCK_SIZE; ++k) {
            tmp += tile_a[threadIdx.y][k] * tile_b[k][threadIdx.x];
        }
        __syncthreads();
    }

    if (row < n && col < n) {
        d_result[row * n + col] = tmp;
    }
}

__global__ void gpu_matrix_transpose(int* mat_in, int* mat_out, unsigned int rows, unsigned int cols) {
    unsigned int idx = blockIdx.x * blockDim.x + threadIdx.x;
    unsigned int idy = blockIdx.y * blockDim.y + threadIdx.y;

    if (idx < cols && idy < rows) {
        unsigned int pos = idy * cols + idx;
        unsigned int trans_pos = idx * rows + idy;
        mat_out[trans_pos] = mat_in[pos];
    }
}

void cpu_matrix_mult(int *h_a, int *h_b, int *h_result, int m, int n, int k) {
    for (int i = 0; i < m; ++i) {
        for (int j = 0; j < k; ++j) {
            int tmp = 0;
            for (int h = 0; h < n; ++h) {
                tmp += h_a[i * n + h] * h_b[h * k + j];
            }
            h_result[i * k + j] = tmp;
        }
    }
}

int main(int argc, char const *argv[]) {
    int m, n, k;
    srand(3333);
    printf("please type in m n and k\n");
    scanf("%d %d %d", &m, &n, &k);

    int *h_a, *h_b, *h_c, *h_cc;
    hipHostMalloc((void **) &h_a, sizeof(int)*m*n, hipHostMallocDefault);
    hipHostMalloc((void **) &h_b, sizeof(int)*n*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_c, sizeof(int)*m*k, hipHostMallocDefault);
    hipHostMalloc((void **) &h_cc, sizeof(int)*m*k, hipHostMallocDefault);

    for (int i = 0; i < m * n; ++i) h_a[i] = rand() % 1024;
    for (int i = 0; i < n * k; ++i) h_b[i] = rand() % 1024;

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);

    int *d_a, *d_b, *d_c;
    hipMalloc((void **) &d_a, sizeof(int)*m*n);
    hipMalloc((void **) &d_b, sizeof(int)*n*k);
    hipMalloc((void **) &d_c, sizeof(int)*m*k);

    hipMemcpy(d_a, h_a, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, sizeof(int)*n*k, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (k + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    if (m == n && n == k) {
        gpu_square_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, n);
    } else {
        gpu_matrix_mult<<<dimGrid, dimBlock>>>(d_a, d_b, d_c, m, n, k);
    }

    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        printf("CUDA kernel launch error: %s\n", hipGetErrorString(err));
        return -1;
    }

    hipMemcpy(h_c, d_c, sizeof(int)*m*k, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on GPU: %f ms.\n\n", m, n, n, k, gpu_elapsed_time_ms);

    hipEventRecord(start, 0);
    cpu_matrix_mult(h_a, h_b, h_cc, m, n, k);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %dx%d . %dx%d on CPU: %f ms.\n\n", m, n, n, k, cpu_elapsed_time_ms);

    int all_ok = 1;
    for (int i = 0; i < m * k; ++i) {
        if (h_cc[i] != h_c[i]) {
            all_ok = 0;
            break;
        }
    }

    if (all_ok) {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    } else {
        printf("incorrect results\n");
    }

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    hipHostFree(h_a);
    hipHostFree(h_b);
    hipHostFree(h_c);
    hipHostFree(h_cc);

    return 0;
}
