#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <sys/time.h>

// Function to get current time in seconds
double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Function to generate a symmetric positive-definite matrix
void generate_spd_matrix(double *A, double *b, int N) {
    int i, j;
    srand(12345);  // Fixed seed for reproducibility
    
    // First generate a random matrix
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            // Generate random value between -0.5 and 0.5
            A[i*N + j] = ((double)rand() / RAND_MAX) - 0.5;
        }
    }
    
    // Make it symmetric: A = 0.5 * (A + A^T)
    for (i = 0; i < N; i++) {
        for (j = 0; j < i; j++) {  // Only need to process lower triangle
            double avg = (A[i*N + j] + A[j*N + i]) * 0.5;
            A[i*N + j] = A[j*N + i] = avg;
        }
    }
    
    // Make it diagonally dominant to ensure positive definiteness
    for (i = 0; i < N; i++) {
        double row_sum = 0.0;
        for (j = 0; j < N; j++) {
            if (i != j) {
                row_sum += fabs(A[i*N + j]);
            }
        }
        // Make diagonal elements larger than sum of other elements in row
        A[i*N + i] = row_sum + 1.0;
    }
    
    // Generate right-hand side vector b
    for (i = 0; i < N; i++) {
        b[i] = ((double)rand() / RAND_MAX) * 10.0;
    }
}

// CUDA kernel for matrix-vector multiplication (y = A*x)
__global__ void matrix_vector_multiply(double *A, double *x, double *y, int N) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < N) {
        double sum = 0.0;
        for (int j = 0; j < N; j++) {
            sum += A[row * N + j] * x[j];
        }
        y[row] = sum;
    }
}

// CUDA kernel for vector addition (a = b + alpha*c)
__global__ void vector_add(double *a, double *b, double *c, double alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        a[idx] = b[idx] + alpha * c[idx];
    }
}

// CUDA kernel for vector subtraction (a = b - alpha*c)
__global__ void vector_subtract(double *a, double *b, double *c, double alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        a[idx] = b[idx] - alpha * c[idx];
    }
}

// CUDA kernel for vector scaling (y = alpha*x)
__global__ void vector_scale(double *y, double *x, double alpha, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (idx < N) {
        y[idx] = alpha * x[idx];
    }
}

// CUDA kernel for dot product (result = sum(a[i]*b[i]))
__global__ void dot_product_kernel(double *a, double *b, double *result, int N) {
    __shared__ double cache[256]; // Assuming block size <= 256
    
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int cacheIndex = threadIdx.x;
    
    double temp = 0.0;
    while (idx < N) {
        temp += a[idx] * b[idx];
        idx += blockDim.x * gridDim.x;
    }
    
    cache[cacheIndex] = temp;
    
    __syncthreads();
    
    // Perform reduction in shared memory
    int i = blockDim.x / 2;
    while (i != 0) {
        if (cacheIndex < i) {
            cache[cacheIndex] += cache[cacheIndex + i];
        }
        __syncthreads();
        i /= 2;
    }
    
    if (cacheIndex == 0) {
        result[blockIdx.x] = cache[0];
    }
}

// Host function for computing dot product
double dot_product(double *d_a, double *d_b, int N, hipblasHandle_t handle) {
    double result;
    hipblasDdot(handle, N, d_a, 1, d_b, 1, &result);
    return result;
}

// Function to verify the solution using CPU
double verify_solution(double *A, double *x, double *b, int N) {
    double *residual = (double*)malloc(N * sizeof(double));
    
    // Compute residual: r = b - A*x
    for (int i = 0; i < N; i++) {
        residual[i] = b[i];
        for (int j = 0; j < N; j++) {
            residual[i] -= A[i*N + j] * x[j];
        }
    }
    
    // Compute norm ||r||
    double norm = 0.0;
    for (int i = 0; i < N; i++) {
        norm += residual[i] * residual[i];
    }
    
    free(residual);
    return sqrt(norm);
}

int main(int argc, char *argv[]) {
    // Default parameters
    int N = 1000;        // Matrix size
    int max_iter = 1000;  // Maximum iterations
    double tol = 1e-6;    // Tolerance
    
    // Parse command line arguments
    if (argc > 1) {
        N = atoi(argv[1]);
    }
    if (argc > 2) {
        max_iter = atoi(argv[2]);
    }
    if (argc > 3) {
        tol = atof(argv[3]);
    }
    
    printf("Conjugate Gradient Method (CUDA)\n");
    printf("Matrix size: %d x %d\n", N, N);
    printf("Maximum iterations: %d\n", max_iter);
    printf("Tolerance: %e\n", tol);
    
    // Allocate host memory
    double *A = (double*)malloc(N * N * sizeof(double));
    double *b = (double*)malloc(N * sizeof(double));
    double *x = (double*)malloc(N * sizeof(double));
    double *x_cublas = (double*)malloc(N * sizeof(double));
    
    if (!A || !b || !x || !x_cublas) {
        printf("Host memory allocation failed\n");
        return 1;
    }
    
    // Generate problem
    generate_spd_matrix(A, b, N);
    
    // Initialize solution
    for (int i = 0; i < N; i++) {
        x[i] = 0.0;
        x_cublas[i] = 0.0;
    }
    
    // Allocate device memory
    double *d_A, *d_b, *d_x, *d_r, *d_p, *d_Ap;
    
    hipMalloc((void**)&d_A, N * N * sizeof(double));
    hipMalloc((void**)&d_b, N * sizeof(double));
    hipMalloc((void**)&d_x, N * sizeof(double));
    hipMalloc((void**)&d_r, N * sizeof(double));
    hipMalloc((void**)&d_p, N * sizeof(double));
    hipMalloc((void**)&d_Ap, N * sizeof(double));
    
    // Copy data to device
    hipMemcpy(d_A, A, N * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice);
    
    // Setup kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    
    // Setup cuBLAS
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Timing variables
    double start_time, end_time;
    double compute_time = 0.0;
    double comm_time = 0.0;
    
    // Variables for CG algorithm
    double alpha, beta;
    double r_dot_r, r_dot_r_new, p_dot_Ap;
    int iter;
    
    // Start timer
    start_time = get_time();
    double compute_start, compute_end, comm_start, comm_end;
    
    // Initialize: r0 = b - A*x0, but since x0 = 0, r0 = b
    comm_start = get_time();
    hipMemcpy(d_r, d_b, N * sizeof(double), hipMemcpyDeviceToDevice);
    // p0 = r0
    hipMemcpy(d_p, d_r, N * sizeof(double), hipMemcpyDeviceToDevice);
    comm_end = get_time();
    comm_time += comm_end - comm_start;
    
    // Initial r_dot_r = r0^T * r0
    compute_start = get_time();
    r_dot_r = dot_product(d_r, d_r, N, handle);
    compute_end = get_time();
    compute_time += compute_end - compute_start;
    
    double initial_residual = sqrt(r_dot_r);
    printf("Initial residual: %e\n", initial_residual);
    
    // Main CG loop
    for (iter = 0; iter < max_iter; iter++) {
        // Compute Ap = A*p
        compute_start = get_time();
        matrix_vector_multiply<<<gridSize, blockSize>>>(d_A, d_p, d_Ap, N);
        hipDeviceSynchronize();
        
        // Compute p_dot_Ap = p^T * Ap
        p_dot_Ap = dot_product(d_p, d_Ap, N, handle);
        
        // Compute alpha = r_dot_r / p_dot_Ap
        alpha = r_dot_r / p_dot_Ap;
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Update x = x + alpha*p
        compute_start = get_time();
        vector_add<<<gridSize, blockSize>>>(d_x, d_x, d_p, alpha, N);
        hipDeviceSynchronize();
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Update r = r - alpha*Ap
        compute_start = get_time();
        vector_subtract<<<gridSize, blockSize>>>(d_r, d_r, d_Ap, alpha, N);
        hipDeviceSynchronize();
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Check convergence
        compute_start = get_time();
        r_dot_r_new = dot_product(d_r, d_r, N, handle);
        
        if (sqrt(r_dot_r_new) < tol * initial_residual) {
            printf("Converged after %d iterations\n", iter + 1);
            break;
        }
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Compute beta = r_dot_r_new / r_dot_r
        compute_start = get_time();
        beta = r_dot_r_new / r_dot_r;
        
        // Update p = r + beta*p
        vector_add<<<gridSize, blockSize>>>(d_p, d_r, d_p, beta, N);
        hipDeviceSynchronize();
        
        // Update r_dot_r for next iteration
        r_dot_r = r_dot_r_new;
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Print progress periodically
        if ((iter + 1) % 100 == 0) {
            printf("Iteration %d: residual = %e\n", iter + 1, sqrt(r_dot_r_new));
        }
    }
    
    // Copy result back to host
    comm_start = get_time();
    hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost);
    comm_end = get_time();
    comm_time += comm_end - comm_start;
    
    // Stop timer
    end_time = get_time();
    double total_time = end_time - start_time;
    
    // Verify solution
    double residual_norm = verify_solution(A, x, b, N);
    
    // Print results
    printf("\n--- Results ---\n");
    printf("Final residual norm: %e\n", residual_norm);
    printf("Iterations: %d\n", iter);
    printf("Total time: %f seconds\n", total_time);
    printf("Compute time: %f seconds\n", compute_time);
    printf("Communication time: %f seconds\n", comm_time);
    printf("Compute/Comm ratio: %f\n", compute_time / comm_time);
    
    // Clean up
    free(A);
    free(b);
    free(x);
    free(x_cublas);
    
    hipFree(d_A);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ap);
    
    hipblasDestroy(handle);
    
    return 0;
}