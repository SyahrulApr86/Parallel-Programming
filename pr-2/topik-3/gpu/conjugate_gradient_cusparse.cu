#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipsparse.h>
#include <sys/time.h>

// Function to get current time in seconds
double get_time() {
    struct timeval tv;
    gettimeofday(&tv, NULL);
    return tv.tv_sec + tv.tv_usec * 1e-6;
}

// Struct for CSR format
typedef struct {
    int *rowPtr;
    int *colIdx;
    double *values;
    int nnz;
} CSRMatrix;

// Function to convert dense matrix to CSR format
void dense_to_csr(double *A, int N, CSRMatrix *csr) {
    // Count non-zero elements
    int nnz = 0;
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (A[i*N + j] != 0.0) {
                nnz++;
            }
        }
    }
    
    // Allocate CSR arrays
    csr->rowPtr = (int*)malloc((N + 1) * sizeof(int));
    csr->colIdx = (int*)malloc(nnz * sizeof(int));
    csr->values = (double*)malloc(nnz * sizeof(double));
    csr->nnz = nnz;
    
    // Fill CSR arrays
    int count = 0;
    csr->rowPtr[0] = 0;
    
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            if (A[i*N + j] != 0.0) {
                csr->colIdx[count] = j;
                csr->values[count] = A[i*N + j];
                count++;
            }
        }
        csr->rowPtr[i + 1] = count;
    }
}

// Function to generate a symmetric positive-definite matrix
void generate_spd_matrix(double *A, double *b, int N) {
    int i, j;
    srand(12345);  // Fixed seed for reproducibility
    
    // First generate a random matrix
    for (i = 0; i < N; i++) {
        for (j = 0; j < N; j++) {
            // Generate random value between -0.5 and 0.5
            A[i*N + j] = ((double)rand() / RAND_MAX) - 0.5;
        }
    }
    
    // Make it symmetric: A = 0.5 * (A + A^T)
    for (i = 0; i < N; i++) {
        for (j = 0; j < i; j++) {  // Only need to process lower triangle
            double avg = (A[i*N + j] + A[j*N + i]) * 0.5;
            A[i*N + j] = A[j*N + i] = avg;
        }
    }
    
    // Make it diagonally dominant to ensure positive definiteness
    for (i = 0; i < N; i++) {
        double row_sum = 0.0;
        for (j = 0; j < N; j++) {
            if (i != j) {
                row_sum += fabs(A[i*N + j]);
            }
        }
        // Make diagonal elements larger than sum of other elements in row
        A[i*N + i] = row_sum + 1.0;
    }
    
    // Generate right-hand side vector b
    for (i = 0; i < N; i++) {
        b[i] = ((double)rand() / RAND_MAX) * 10.0;
    }
}

// Function to verify the solution using CPU
double verify_solution(double *A, double *x, double *b, int N) {
    double *residual = (double*)malloc(N * sizeof(double));
    
    // Compute residual: r = b - A*x
    for (int i = 0; i < N; i++) {
        residual[i] = b[i];
        for (int j = 0; j < N; j++) {
            residual[i] -= A[i*N + j] * x[j];
        }
    }
    
    // Compute norm ||r||
    double norm = 0.0;
    for (int i = 0; i < N; i++) {
        norm += residual[i] * residual[i];
    }
    
    free(residual);
    return sqrt(norm);
}

// Generic CUDA error checking function
#define CHECK_CUDA_ERROR(val) check_cuda((val), #val, __FILE__, __LINE__)
inline void check_cuda(hipError_t result, char const *const func, const char *const file, int const line) {
    if (result) {
        fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\"\n",
                file, line, static_cast<unsigned int>(result), hipGetErrorString(result), func);
        exit(EXIT_FAILURE);
    }
}

// Custom sparse matrix-vector multiplication kernel
__global__ void spmv_csr_kernel(int num_rows, int* row_ptrs, int* col_indices, 
                              double* values, double* x, double* y) {
    int row = blockIdx.x * blockDim.x + threadIdx.x;
    
    if (row < num_rows) {
        double dot = 0.0;
        int row_start = row_ptrs[row];
        int row_end = row_ptrs[row + 1];
        
        for (int i = row_start; i < row_end; i++) {
            dot += values[i] * x[col_indices[i]];
        }
        
        y[row] = dot;
    }
}

int main(int argc, char *argv[]) {
    // Default parameters
    int N = 1000;        // Matrix size
    int max_iter = 1000;  // Maximum iterations
    double tol = 1e-6;    // Tolerance
    
    // Parse command line arguments
    if (argc > 1) {
        N = atoi(argv[1]);
    }
    if (argc > 2) {
        max_iter = atoi(argv[2]);
    }
    if (argc > 3) {
        tol = atof(argv[3]);
    }
    
    printf("Conjugate Gradient Method (cuSPARSE)\n");
    printf("Matrix size: %d x %d\n", N, N);
    printf("Maximum iterations: %d\n", max_iter);
    printf("Tolerance: %e\n", tol);
    
    // Allocate host memory
    double *A = (double*)malloc(N * N * sizeof(double));
    double *b = (double*)malloc(N * sizeof(double));
    double *x = (double*)malloc(N * sizeof(double));
    
    if (!A || !b || !x) {
        printf("Host memory allocation failed\n");
        return 1;
    }
    
    // Generate problem
    generate_spd_matrix(A, b, N);
    
    // Initialize solution
    for (int i = 0; i < N; i++) {
        x[i] = 0.0;
    }
    
    // Convert to CSR format
    CSRMatrix csr;
    dense_to_csr(A, N, &csr);
    
    printf("Matrix has %d non-zero elements (%.2f%% sparsity)\n", 
           csr.nnz, 100.0 * (1.0 - (double)csr.nnz / (N * N)));
    
    // Allocate device memory
    double *d_b, *d_x, *d_r, *d_p, *d_Ap;
    int *d_csrRowPtr, *d_csrColIdx;
    double *d_csrValues;
    
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_csrRowPtr, (N + 1) * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_csrColIdx, csr.nnz * sizeof(int)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_csrValues, csr.nnz * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_b, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_x, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_r, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_p, N * sizeof(double)));
    CHECK_CUDA_ERROR(hipMalloc((void**)&d_Ap, N * sizeof(double)));
    
    // Copy data to device
    CHECK_CUDA_ERROR(hipMemcpy(d_csrRowPtr, csr.rowPtr, (N + 1) * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_csrColIdx, csr.colIdx, csr.nnz * sizeof(int), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_csrValues, csr.values, csr.nnz * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_b, b, N * sizeof(double), hipMemcpyHostToDevice));
    CHECK_CUDA_ERROR(hipMemcpy(d_x, x, N * sizeof(double), hipMemcpyHostToDevice));
    
    // Setup cuBLAS
    hipblasHandle_t cublasHandle;
    hipblasCreate(&cublasHandle);
    
    // Setup kernel launch parameters
    int blockSize = 256;
    int gridSize = (N + blockSize - 1) / blockSize;
    
    // Timing variables
    double start_time, end_time;
    double compute_time = 0.0;
    double comm_time = 0.0;
    
    // Variables for CG algorithm
    double alpha, beta;
    double r_dot_r, r_dot_r_new, p_dot_Ap;
    int iter;
    
    // Start timer
    start_time = get_time();
    double compute_start, compute_end, comm_start, comm_end;
    
    // Initialize: r0 = b - A*x0, but since x0 = 0, r0 = b
    comm_start = get_time();
    CHECK_CUDA_ERROR(hipMemcpy(d_r, d_b, N * sizeof(double), hipMemcpyDeviceToDevice));
    // p0 = r0
    CHECK_CUDA_ERROR(hipMemcpy(d_p, d_r, N * sizeof(double), hipMemcpyDeviceToDevice));
    comm_end = get_time();
    comm_time += comm_end - comm_start;
    
    // Initial r_dot_r = r0^T * r0
    compute_start = get_time();
    hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r_dot_r);
    compute_end = get_time();
    compute_time += compute_end - compute_start;
    
    double initial_residual = sqrt(r_dot_r);
    printf("Initial residual: %e\n", initial_residual);
    
    // Main CG loop
    for (iter = 0; iter < max_iter; iter++) {
        // Compute Ap = A*p using our custom SPMV kernel
        compute_start = get_time();
        
        // Use custom kernel for sparse matrix-vector multiplication
        spmv_csr_kernel<<<gridSize, blockSize>>>(N, d_csrRowPtr, d_csrColIdx, d_csrValues, d_p, d_Ap);
        CHECK_CUDA_ERROR(hipDeviceSynchronize());
        
        // Compute p_dot_Ap = p^T * Ap
        hipblasDdot(cublasHandle, N, d_p, 1, d_Ap, 1, &p_dot_Ap);
        
        // Compute alpha = r_dot_r / p_dot_Ap
        alpha = r_dot_r / p_dot_Ap;
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Update x = x + alpha*p
        compute_start = get_time();
        hipblasDaxpy(cublasHandle, N, &alpha, d_p, 1, d_x, 1);
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Update r = r - alpha*Ap
        compute_start = get_time();
        double neg_alpha = -alpha;
        hipblasDaxpy(cublasHandle, N, &neg_alpha, d_Ap, 1, d_r, 1);
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Check convergence
        compute_start = get_time();
        hipblasDdot(cublasHandle, N, d_r, 1, d_r, 1, &r_dot_r_new);
        
        if (sqrt(r_dot_r_new) < tol * initial_residual) {
            printf("Converged after %d iterations\n", iter + 1);
            break;
        }
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Compute beta = r_dot_r_new / r_dot_r
        compute_start = get_time();
        beta = r_dot_r_new / r_dot_r;
        
        // Update p = r + beta*p (first scale p by beta, then add r)
        hipblasDscal(cublasHandle, N, &beta, d_p, 1);
        double one = 1.0;
        hipblasDaxpy(cublasHandle, N, &one, d_r, 1, d_p, 1);
        
        // Update r_dot_r for next iteration
        r_dot_r = r_dot_r_new;
        compute_end = get_time();
        compute_time += compute_end - compute_start;
        
        // Print progress periodically
        if ((iter + 1) % 100 == 0) {
            printf("Iteration %d: residual = %e\n", iter + 1, sqrt(r_dot_r_new));
        }
    }
    
    // Copy result back to host
    comm_start = get_time();
    CHECK_CUDA_ERROR(hipMemcpy(x, d_x, N * sizeof(double), hipMemcpyDeviceToHost));
    comm_end = get_time();
    comm_time += comm_end - comm_start;
    
    // Stop timer
    end_time = get_time();
    double total_time = end_time - start_time;
    
    // Verify solution
    double residual_norm = verify_solution(A, x, b, N);
    
    // Print results
    printf("\n--- Results ---\n");
    printf("Final residual norm: %e\n", residual_norm);
    printf("Iterations: %d\n", iter);
    printf("Total time: %f seconds\n", total_time);
    printf("Compute time: %f seconds\n", compute_time);
    printf("Communication time: %f seconds\n", comm_time);
    printf("Compute/Comm ratio: %f\n", compute_time / comm_time);
    
    // Clean up
    free(A);
    free(b);
    free(x);
    free(csr.rowPtr);
    free(csr.colIdx);
    free(csr.values);
    
    hipFree(d_csrRowPtr);
    hipFree(d_csrColIdx);
    hipFree(d_csrValues);
    hipFree(d_b);
    hipFree(d_x);
    hipFree(d_r);
    hipFree(d_p);
    hipFree(d_Ap);
    
    hipblasDestroy(cublasHandle);
    
    return 0;
}